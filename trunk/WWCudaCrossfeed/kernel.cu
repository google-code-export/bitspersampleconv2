// 日本語

#include "Util.h"
#include "CrossfeedF.h"

#include <stdio.h>
#include <string.h> //< memset()
#include <math.h>
#include <assert.h>

#include <vector>
#include <float.h>

int wmain(int argc, wchar_t *argv[])
{
    int result = 1;
    int ercd;
    int id = -1;
    size_t nFFT;
    CrossfeedParam crossfeedParam;
    WWFlacMetadata meta;
    uint8_t * picture = NULL;
    hipfftComplex * inPcmSpectra[PCT_NUM];

    std::vector<PcmSamplesPerChannel> pcmSamples;

    if (argc != 4) {
        printf("Usage: %S coeffFile inputFile outputFile\n", argv[0]);
        goto END;
    }

    if (!ReadCrossfeeedParamsFromFileF(argv[1], &crossfeedParam)) {
        printf("Error: could not read crossfeed param file %S\n", argv[1]);
        goto END;
    }

    id = WWFlacRW_DecodeAll(argv[2]);
    if (id < 0) {
        printf("Error: Read failed %S\n", argv[2]);
        goto END;
    }

    ercd = WWFlacRW_GetDecodedMetadata(id, meta);
    if (ercd < 0) {
        printf("Error: Read meta failed %S\n", argv[2]);
        goto END;
    }

    if (0 < meta.pictureBytes) {
        picture = new uint8_t[meta.pictureBytes];
        ercd = WWFlacRW_GetDecodedPicture(id, picture, meta.pictureBytes);
        if (ercd < 0) {
            printf("Error: Read meta failed %S\n", argv[2]);
            goto END;
        }
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: channel count mismatch. FLAC ch=%d, crossfeed ch=%d\n", meta.channels, crossfeedParam.numChannels);
        goto END;
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: samplerate mismatch. FLAC=%d, crossfeed=%d\n", meta.sampleRate, crossfeedParam.sampleRate);
        goto END;
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        size_t bytes = (size_t)(meta.totalSamples * (meta.bitsPerSample/8));
        uint8_t *buff = new uint8_t[bytes];
        WWFlacRW_GetDecodedPcmBytes(id, ch, 0, buff, bytes);

        PcmSamplesPerChannel ppc;
        ppc.Init();
        ppc.totalSamples = (size_t)meta.totalSamples;
        ppc.inputPcm = new float[(size_t)(meta.totalSamples * sizeof(float))];
        SetInputPcmSamplesF(buff, meta.bitsPerSample, &ppc);

        delete [] buff;
        buff = NULL;

        {
            // 低音域
            PcmSamplesPerChannel lowFreq;
            lowFreq.Init();
            lowFreq.totalSamples = ppc.totalSamples;
            lowFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilterF(gLpf, sizeof gLpf/sizeof gLpf[0], ppc, &lowFreq)) {
                goto END;
            }
            pcmSamples.push_back(lowFreq);
        }

        {
            // 高音域
            PcmSamplesPerChannel highFreq;
            highFreq.Init();
            highFreq.totalSamples = ppc.totalSamples;
            highFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilterF(gHpf, sizeof gHpf/sizeof gHpf[0], ppc, &highFreq)) {
                goto END;
            }
            pcmSamples.push_back(highFreq);
        }
        ppc.Term();
    }

    WWFlacRW_DecodeEnd(id);
    id = -1;

    nFFT = (size_t)((crossfeedParam.coeffSize < meta.totalSamples) ? meta.totalSamples : crossfeedParam.coeffSize);
    nFFT = NextPowerOf2(nFFT);

    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        crossfeedParam.spectra[i] = CreateSpectrumF(crossfeedParam.coeffs[i], crossfeedParam.coeffSize, nFFT);
        if (crossfeedParam.spectra[i] == NULL) {
            goto END;
        }
        crossfeedParam.fftSize = nFFT;
    }
    for (int i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].spectrum = CreateSpectrumF(pcmSamples[i].inputPcm, pcmSamples[i].totalSamples, nFFT);
        if (pcmSamples[i].spectrum == NULL) {
            goto END;
        }
        pcmSamples[i].fftSize = nFFT;
        inPcmSpectra[i] = pcmSamples[i].spectrum;
    }

    pcmSamples[0].outputPcm = CrossfeedMixF(inPcmSpectra,
            &crossfeedParam.spectra[0], &crossfeedParam.spectra[4], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[0].outputPcm == NULL) {
        goto END;
    }
    pcmSamples[1].outputPcm = CrossfeedMixF(inPcmSpectra,
            &crossfeedParam.spectra[2], &crossfeedParam.spectra[6], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[1].outputPcm == NULL) {
        goto END;
    }

    NormalizeOutputPcmF(pcmSamples);

    // 出力bit depth == 24bit
    meta.bitsPerSample = 24;
    if (!WriteFlacFileF(meta, picture, pcmSamples, argv[3])) {
        printf("Error: WriteFlac(%S) failed\n", argv[3]);
        goto END;
    }

    result = 0;

END:
    delete [] picture;
    picture = NULL;

    for (size_t i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].Term();
    }
    pcmSamples.clear();

    crossfeedParam.Term();

    if (result != 0) {
        printf("Failed!\n");
    } else {
        printf("    maximum used CUDA memory: %lld Mbytes\n", gCudaMaxBytes / 1024/ 1024);
        printf("Succeeded to write %S.\n", argv[3]);
        assert(gCudaAllocatedBytes == 0);
    }

    return result;
}