#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h> //< memset()
#include <math.h>
#include <assert.h>

#include <hipfft/hipfft.h>

#include "WWFlacRW.h"
#include <vector>
#include <float.h>

#define CROSSFEED_COEF_NUM (8)
#define NUM_THREADS_PER_BLOCK (32)
#define BLOCK_X (32768)

enum PcmChannelType {
    PCT_LeftLow,
    PCT_LeftHigh,
    PCT_RightLow,
    PCT_RightHigh,
    PCT_NUM
};

// 44.1kHz�p 1kHz�ȉ������o��LPF�B�f�B���C��HPF�ƍ��킹��B
static float gLpf[] = {
    0.0, 0.0, 0.0, 0.0,
    0.0, 0.0, 0.0, 0.0,
    0.005228327, 0.003249754, 0.004192373, 0.005265026,
    0.006468574, 0.007797099, 0.009237486, 0.010779043,
    0.012417001, 0.014132141, 0.01589555, 0.017701121,
    0.019508703, 0.021304869, 0.023059883,0.024747905,
    0.02634363, 0.027823228, 0.029158971, 0.030331066,
    0.031319484, 0.032104039, 0.032676435, 0.033022636,
    0.033138738, 0.033022636, 0.032676435, 0.032104039,
    0.031319484, 0.030331066, 0.029158971, 0.027823228,
    0.02634363, 0.024747905, 0.023059883, 0.021304869,
    0.019508703, 0.017701121, 0.01589555, 0.014132141,
    0.012417001, 0.010779043, 0.009237486, 0.007797099,
    0.006468574, 0.005265026, 0.004192373, 0.003249754,
    0.005228327 };

// 44.1kHz�p 1kHz�ȏ�����o��HPF�BLPF�ƃR���v�������^���[�ɂȂ��Ă���B
static float gHpf[] = {
            0.18284994666072, 0.135236586884136, 0.122587729966973, 0.0194160387180315,
            -0.018554433938607, -0.147953150443285, -0.151175453402614, -0.349031089410704,
            0.402427310594025, -0.0290473999611028, -0.158700363343139, -0.0615684548751206,
            -0.0757049588858041, -0.00276365388846877, -0.0170037945344803, 0.024517130588635,

            -0.00946482719827604, 0.0131620130045816, -0.0290320503133136, 0.00288266370987212,
            -0.0316496185674791, 0.0377990630692439, 0.00347129302294323, 0.190514306269757,
            -0.0464892033979637, -0.0815022375716482, 0.0915280717216996, 0.0379499306403254,
            0.0878060905218899, 0.0380322544842594, 0.0621210160273231, 0.0199502643669319,
            
            0.0479758154442058, 0.0199502643669309, 0.062121016027322, 0.0380322544842602,
            0.0878060905218885, 0.0379499306403286, 0.0915280717216994, -0.0815022375716453,
            -0.0464892033979671, 0.190514306269758, 0.00347129302294125, 0.0377990630692462,
            -0.0316496185674813, 0.0028826637098741, -0.029032050313314, 0.0131620130045817,

            -0.00946482719827875, 0.024517130588635, -0.0170037945344796, -0.00276365388846837,
            -0.0757049588858039, -0.061568454875121, -0.158700363343138, -0.0290473999611048,
            0.402427310594025, -0.349031089410705, -0.151175453402611, -0.147953150443284,
            -0.0185544339386063, 0.0194160387180304, 0.122587729966972, 0.135236586884135,
            0.18284994666072, };

struct CrossfeedParam {
    int numChannels;
    float *coeffs[CROSSFEED_COEF_NUM];
    hipfftComplex *spectra[CROSSFEED_COEF_NUM];

    int sampleRate;
    int coeffSize;

    CrossfeedParam(void) {
        numChannels = 0;
        sampleRate = 0;
        coeffSize = 0;

        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            coeffs[i]  = NULL;
            spectra[i] = NULL;
        }
    }

    void Term(void) {
        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            delete [] coeffs[i];
            coeffs[i] = NULL;

            hipFree(spectra[i]);
            spectra[i] = NULL;
        }
    }
};

struct PcmSamplesPerChannel {
    size_t totalSamples;
    float *inputPcm;
    float *outputPcm;
    hipfftComplex *spectrum;

    void Init(void) {
        inputPcm = NULL;
        outputPcm = NULL;
        spectrum = NULL;
    }

    void Term(void) {
        delete [] inputPcm;
        inputPcm = NULL;

        delete [] outputPcm;
        outputPcm = NULL;

        hipFree(spectrum);
        spectrum = NULL;
    }
};

static bool
ReadOneLine(FILE *fp, char *line_return, size_t lineBytes)
{
    line_return[0] = 0;
    int c;
    int pos = 0;

    do {
        c = fgetc(fp);
        if (c == EOF || c == '\n') {
            break;
        }

        if (c != '\r') {
            line_return[pos] = (char)c;
            line_return[pos+1] = 0;
            ++pos;
        }
    } while (c != EOF && pos < (int)lineBytes -1);

    return c != EOF;
}

#define CHECKED(x) if (!(x)) { goto END; }

static bool
ReadCrossfeeedParamsFromFile(const wchar_t *path, CrossfeedParam *param_return)
{
    assert(param_return);

    char buff[256];
    bool result = false;
    FILE *fp;
    errno_t ercd = _wfopen_s(&fp, path, L"rb");
    if (NULL == fp || 0 != ercd) {
        return false;
    }

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    CHECKED(0 == strncmp(buff, "CFD2", 4));

    param_return->numChannels = 2;

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->sampleRate);

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->coeffSize);

    CHECKED(0 < param_return->coeffSize);

    // �R�����g�s�B�X�L�b�v����B
    CHECKED(ReadOneLine(fp, buff, sizeof buff));

    for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
        param_return->coeffs[ch] = new float[param_return->coeffSize];
    }

    for (int i=0; i<param_return->coeffSize; ++i) {
#if CROSSFEED_COEF_NUM != 8
#  error
#endif
        double v[CROSSFEED_COEF_NUM];

        CHECKED(ReadOneLine(fp, buff, sizeof buff));
        CHECKED(8 == sscanf(buff, "%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf", &v[0], &v[1], &v[2], &v[3], &v[4], &v[5], &v[6], &v[7]));

        for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
            param_return->coeffs[ch][i] = (float)v[ch];
        }
    }

    result = true;

END:
    fclose(fp);
    fp = NULL;
    return result;
}

static void
SetInputPcmSamples(uint8_t *buff, int bitsPerSample, PcmSamplesPerChannel *ppc_return)
{
    assert(ppc_return);

    switch (bitsPerSample) {
    case 16:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            short v = (short)(buff[samplePos*2] + (buff[samplePos*2+1]<<8));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 32768.0f);
        }
        break;
    case 24:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            int v = (int)((buff[samplePos*3]<<8) + (buff[samplePos*3+1]<<16) + (buff[samplePos*3+2]<<24));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 2147483648.0f);
        }
        break;
    default:
        assert(!"not supported");
        break;
    }
}

static size_t
NextPowerOf2(size_t v)
{
    size_t result = 1;
    if (INT_MAX+1U < v) {
        printf("Error: NextPowerOf2(%d) too large!\n", v);
        return 0;
    }
    while (result < v) {
        result *= 2;
    }
    return result;
}

static const char *
CudaFftGetErrorString(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS:       return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:  return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:  return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:  return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:    return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:   return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:   return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";

        case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE:            return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR:               return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE:              return "HIPFFT_NO_WORKSPACE";
        default: return "unknown";
    }
}


#define CHK_CUDAERROR(x)                                                              \
    ercd = x;                                                                         \
    if (hipSuccess != ercd) {                                                        \
        printf("%s failed. errorcode=%d (%s)\n", #x, ercd, hipGetErrorString(ercd)); \
        return NULL;                                                                  \
    }

#define CHK_CUFFT(x)                                                                               \
    fftResult = x;                                                                                 \
    if (hipSuccess != fftResult) {                                                                \
        printf("%s failed. errorcode=%d (%s)\n", #x, fftResult, CudaFftGetErrorString(fftResult)); \
        return NULL;                                                                               \
    }

__global__ void
ElementWiseMulCuda(hipfftComplex *C, hipfftComplex *A, hipfftComplex *B)
{
    int offs = threadIdx.x + NUM_THREADS_PER_BLOCK * (blockIdx.x + BLOCK_X * blockIdx.y);
    C[offs].x = A[offs].x * B[offs].x - A[offs].y * B[offs].y;
    C[offs].y = A[offs].x * B[offs].y + A[offs].y * B[offs].x;
}

__global__ void
ElementWiseAddCuda(hipfftReal *C, hipfftReal *A, hipfftReal *B)
{
    int offs = threadIdx.x + NUM_THREADS_PER_BLOCK * (blockIdx.x + BLOCK_X * blockIdx.y);
    C[offs] = A[offs] + B[offs];
}

static void
CudaElementWiseMul(int count, hipfftComplex *dest, hipfftComplex *from0, hipfftComplex *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    if ((count / NUM_THREADS_PER_BLOCK) <= 1) {
        threads.x = count;
    } else {
        threads.x = NUM_THREADS_PER_BLOCK;
        threads.y = 1;
        threads.z = 1;
        int countRemain = count / NUM_THREADS_PER_BLOCK;
        if ((countRemain / BLOCK_X) <= 1) {
            blocks.x = countRemain;
            blocks.y = 1;
            blocks.z = 1;
        } else {
            blocks.x = BLOCK_X;
            countRemain /= BLOCK_X;
            blocks.y = countRemain;
            blocks.z = 1;
        }
    }

    ElementWiseMulCuda<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static void
CudaElementWiseAdd(int count, hipfftReal *dest, hipfftReal *from0, hipfftReal *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    if ((count / NUM_THREADS_PER_BLOCK) <= 1) {
        threads.x = count;
    } else {
        threads.x = NUM_THREADS_PER_BLOCK;
        threads.y = 1;
        threads.z = 1;
        int countRemain = count / NUM_THREADS_PER_BLOCK;
        if ((countRemain / BLOCK_X) <= 1) {
            blocks.x = countRemain;
            blocks.y = 1;
            blocks.z = 1;
        } else {
            blocks.x = BLOCK_X;
            countRemain /= BLOCK_X;
            blocks.y = countRemain;
            blocks.z = 1;
        }
    }

    ElementWiseAddCuda<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static hipfftComplex *
CreateSpectrum(float *timeDomainData, int numSamples, int fftSize)
{
    hipfftReal *cuFromT = NULL;
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftComplex *spectrum;
    hipfftHandle plan = 0;

    CHK_CUDAERROR(hipMalloc((void**)&cuFromT, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemset((void*)cuFromT, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(cuFromT, timeDomainData, numSamples * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAERROR(hipMalloc((void**)&spectrum, sizeof(hipfftComplex)*fftSize));

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, cuFromT, spectrum));

    hipDeviceSynchronize();

    hipfftDestroy(plan);
    plan = 0;

    hipFree(cuFromT);
    cuFromT = NULL;

    return spectrum;
}

static float *
FirFilter(float *firCoeff, size_t firCoeffNum, PcmSamplesPerChannel &input, PcmSamplesPerChannel *pOutput)
{
    size_t fftSize = (firCoeffNum < input.totalSamples) ? input.totalSamples: firCoeffNum;
    fftSize = NextPowerOf2(fftSize);
    if (fftSize == 0) {
        return NULL;
    }

    hipError_t ercd;
    hipfftResult fftResult;
    hipfftReal *coefTime = NULL;
    hipfftReal *pcmTime = NULL;
    hipfftReal *resultTime = NULL;
    hipfftComplex *coefFreq = NULL;
    hipfftComplex *pcmFreq = NULL;
    hipfftComplex *resultFreq = NULL;
    hipfftHandle plan = 0;

    CHK_CUDAERROR(hipMalloc((void**)&coefTime, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemset((void*)coefTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(coefTime, firCoeff, firCoeffNum * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAERROR(hipMalloc((void**)&coefFreq, sizeof(hipfftComplex)*fftSize));

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, coefTime, coefFreq));

    hipDeviceSynchronize();

    hipfftDestroy(plan);
    plan = 0;

    hipFree(coefTime);
    coefTime = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&pcmTime, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemset((void*)pcmTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(pcmTime, input.inputPcm, input.totalSamples * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAERROR(hipMalloc((void**)&pcmFreq, sizeof(hipfftComplex)*fftSize));

    hipDeviceSynchronize();

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, pcmTime, pcmFreq));

    hipDeviceSynchronize();

    hipfftDestroy(plan);
    plan = 0;

    hipFree(pcmTime);
    pcmTime = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&resultFreq, sizeof(hipfftComplex)*fftSize));
    CudaElementWiseMul(fftSize, resultFreq, coefFreq, pcmFreq);

    hipDeviceSynchronize();

    hipFree(coefFreq);
    coefFreq = NULL;
    hipFree(pcmFreq);
    pcmFreq = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&resultTime, sizeof(hipfftReal)*fftSize));

    hipDeviceSynchronize();

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_C2R, 1));
    CHK_CUFFT(hipfftExecC2R(plan, resultFreq, resultTime));

    hipDeviceSynchronize();

    hipfftDestroy(plan);
    plan = 0;

    hipFree(resultFreq);
    resultFreq = NULL;

    CHK_CUDAERROR(hipMemcpy(pOutput->inputPcm, resultTime, input.totalSamples * sizeof(float), hipMemcpyDeviceToHost));
    
    hipDeviceSynchronize();

    hipFree(resultTime);
    resultTime = NULL;

    return pOutput->inputPcm;
}

static float *
CrossfeedMix(hipfftComplex *inPcmSpectra[PCT_NUM], hipfftComplex *coeffLo[2], hipfftComplex *coeffHi[2], int nFFT, int pcmSamples)
{
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftHandle plan = 0;
    hipfftComplex *cuFreq = NULL;
    hipfftReal *cuTime[PCT_NUM] = {NULL, NULL, NULL, NULL};
    hipfftReal *cuTimeMixedLo = NULL;
    hipfftReal *cuTimeMixedHi = NULL;
    hipfftReal *cuTimeMixed = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&cuFreq,      sizeof(hipfftComplex)*nFFT));

    hipDeviceSynchronize();

    CHK_CUFFT(hipfftPlan1d(&plan, nFFT, HIPFFT_C2R, 1));

    for (int ch=0; ch<2; ++ch) {
        CudaElementWiseMul(nFFT, cuFreq, inPcmSpectra[ch*2], coeffLo[ch]);
    
        hipDeviceSynchronize();

        CHK_CUDAERROR(hipMalloc((void**)&cuTime[ch*2], sizeof(hipfftReal)*nFFT));
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2]));

        hipDeviceSynchronize();

        CudaElementWiseMul(nFFT, cuFreq, inPcmSpectra[ch*2+1], coeffHi[ch]);
    
        hipDeviceSynchronize();

        CHK_CUDAERROR(hipMalloc((void**)&cuTime[ch*2+1], sizeof(hipfftReal)*nFFT));
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2+1]));

        hipDeviceSynchronize();
    }

    hipfftDestroy(plan);
    plan = 0;

    hipFree(cuFreq);
    cuFreq = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&cuTimeMixedLo, sizeof(hipfftReal)*nFFT));
    CHK_CUDAERROR(hipMalloc((void**)&cuTimeMixedHi, sizeof(hipfftReal)*nFFT));
    CHK_CUDAERROR(hipMalloc((void**)&cuTimeMixed, sizeof(hipfftReal)*nFFT));

    hipDeviceSynchronize();

    CudaElementWiseAdd(nFFT, cuTimeMixedLo, cuTime[0], cuTime[2]);
    CudaElementWiseAdd(nFFT, cuTimeMixedHi, cuTime[1], cuTime[3]);
    CudaElementWiseAdd(nFFT, cuTimeMixed, cuTimeMixedLo, cuTimeMixedHi);

    for (int i=0; i<PCT_NUM; ++i) {
        hipFree(cuTime[i]);
        cuTime[i] = NULL;
    }
    hipFree(cuTimeMixedLo);
    cuTimeMixedLo = NULL;
    hipFree(cuTimeMixedHi);
    cuTimeMixedHi = NULL;

    hipDeviceSynchronize();

    float *result = new float[pcmSamples];
    CHK_CUDAERROR(hipMemcpy(result, cuTimeMixed, pcmSamples * sizeof(float), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(cuTimeMixed);
    cuTimeMixed = NULL;

    hipDeviceSynchronize();

    return result;
}

static void
NormalizeOutputPcm(std::vector<PcmSamplesPerChannel> & pcmSamples)
{
    float minV = FLT_MAX;
    float maxV = FLT_MIN;

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }

        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            if (maxV < pcmSamples[ch].outputPcm[i]) {
                maxV = pcmSamples[ch].outputPcm[i];
            }
            if (pcmSamples[ch].outputPcm[i] < minV) {
                minV = pcmSamples[ch].outputPcm[i];
            }
        }
    }

    float absMax = (fabsf(minV) < fabsf(maxV)) ? fabsf(maxV) : fabsf(minV);
    float scale = 1.0f;
    if ((8388607.0f / 8388608.0f) < absMax) {
        scale = (8388607.0f / 8388608.0f) / absMax;
    }

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            pcmSamples[ch].outputPcm[i] *= scale;
        }
    }
}

static bool
WriteFlacFile(const WWFlacMetadata &meta, const uint8_t *picture, std::vector<PcmSamplesPerChannel> &pcmSamples, const wchar_t *path)
{
    bool result = false;
    int rv;
    int pictureBytes = meta.pictureBytes;

    int id = WWFlacRW_EncodeInit(meta);
    if (id < 0) {
        return false;
    }

    if (0 < pictureBytes) {
        rv = WWFlacRW_EncodeSetPicture(id, picture, pictureBytes);
        if (rv < 0) {
            goto END;
        }
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        uint8_t *pcmDataUint8 = new uint8_t[(size_t)(meta.totalSamples * 3)];
        for (int i=0; i<meta.totalSamples; ++i) {
            int v = (int)(8388608.0f * pcmSamples[ch].outputPcm[i]);
            pcmDataUint8[i*3+0] = v&0xff;
            pcmDataUint8[i*3+1] = (v>>8)&0xff;
            pcmDataUint8[i*3+2] = (v>>16)&0xff;
        }

        rv = WWFlacRW_EncodeAddPcm(id, ch, pcmDataUint8, meta.totalSamples*3);
        if (rv < 0) {
            goto END;
        }
        delete [] pcmDataUint8;
        pcmDataUint8 = NULL;
    }

    rv = WWFlacRW_EncodeRun(id, path);
    if (rv < 0) {
        goto END;
    }

    result = true;
END:

    WWFlacRW_EncodeEnd(id);

    return result;
}

int wmain(int argc, wchar_t *argv[])
{
    int result = 1;
    int ercd;
    int id = -1;
    size_t nFFT;
    CrossfeedParam crossfeedParam;
    WWFlacMetadata meta;
    uint8_t * picture = NULL;
    hipfftComplex * inPcmSpectra[PCT_NUM];
    int64_t usedGpuMemoryBytes = 0;

    std::vector<PcmSamplesPerChannel> pcmSamples;

    if (argc != 4) {
        printf("Usage: %S coeffFile inputFile outputFile\n", argv[0]);
        goto END;
    }

    if (!ReadCrossfeeedParamsFromFile(argv[1], &crossfeedParam)) {
        printf("Error: could not read crossfeed param file %S\n", argv[1]);
        goto END;
    }

    id = WWFlacRW_DecodeAll(argv[2]);
    if (id < 0) {
        printf("Error: Read failed %S\n", argv[2]);
        goto END;
    }

    ercd = WWFlacRW_GetDecodedMetadata(id, meta);
    if (ercd < 0) {
        printf("Error: Read meta failed %S\n", argv[2]);
        goto END;
    }

    if (0 < meta.pictureBytes) {
        picture = new uint8_t[meta.pictureBytes];
        ercd = WWFlacRW_GetDecodedPicture(id, picture, meta.pictureBytes);
        if (ercd < 0) {
            printf("Error: Read meta failed %S\n", argv[2]);
            goto END;
        }
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: channel count mismatch. FLAC ch=%d, crossfeed ch=%d\n", meta.channels, crossfeedParam.numChannels);
        goto END;
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: samplerate mismatch. FLAC=%d, crossfeed=%d\n", meta.sampleRate, crossfeedParam.sampleRate);
        goto END;
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        size_t bytes = (size_t)(meta.totalSamples * (meta.bitsPerSample/8));
        uint8_t *buff = new uint8_t[bytes];
        WWFlacRW_GetDecodedPcmBytes(id, ch, 0, buff, bytes);

        PcmSamplesPerChannel ppc;
        ppc.Init();
        ppc.totalSamples = (size_t)meta.totalSamples;
        ppc.inputPcm = new float[(size_t)(meta.totalSamples * sizeof(float))];
        SetInputPcmSamples(buff, meta.bitsPerSample, &ppc);

        delete [] buff;
        buff = NULL;

        {
            // �ቹ��
            PcmSamplesPerChannel lowFreq;
            lowFreq.Init();
            lowFreq.totalSamples = ppc.totalSamples;
            lowFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilter(gLpf, sizeof gLpf/sizeof gLpf[0], ppc, &lowFreq)) {
                goto END;
            }
            pcmSamples.push_back(lowFreq);
        }

        {
            // ������
            PcmSamplesPerChannel highFreq;
            highFreq.Init();
            highFreq.totalSamples = ppc.totalSamples;
            highFreq.inputPcm = new float[ppc.totalSamples];
            if (NULL == FirFilter(gHpf, sizeof gHpf/sizeof gHpf[0], ppc, &highFreq)) {
                goto END;
            }
            pcmSamples.push_back(highFreq);
        }
        ppc.Term();
    }

    WWFlacRW_DecodeEnd(id);
    id = -1;

    nFFT = (size_t)((crossfeedParam.coeffSize < meta.totalSamples) ? meta.totalSamples : crossfeedParam.coeffSize);
    nFFT = NextPowerOf2(nFFT);

    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        crossfeedParam.spectra[i] = CreateSpectrum(crossfeedParam.coeffs[i], crossfeedParam.coeffSize, nFFT);
        if (crossfeedParam.spectra[i] == NULL) {
            goto END;
        }
        usedGpuMemoryBytes += nFFT * sizeof(hipfftComplex);
    }
    for (int i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].spectrum = CreateSpectrum(pcmSamples[i].inputPcm, pcmSamples[i].totalSamples, nFFT);
        if (pcmSamples[i].spectrum == NULL) {
            goto END;
        }
        usedGpuMemoryBytes += nFFT * sizeof(hipfftComplex);
        inPcmSpectra[i] = pcmSamples[i].spectrum;
    }

    pcmSamples[0].outputPcm = CrossfeedMix(inPcmSpectra, &crossfeedParam.spectra[0], &crossfeedParam.spectra[4], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[0].outputPcm == NULL) {
        usedGpuMemoryBytes += nFFT * sizeof(hipfftReal);
        goto END;
    }
    pcmSamples[1].outputPcm = CrossfeedMix(inPcmSpectra, &crossfeedParam.spectra[2], &crossfeedParam.spectra[6], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[1].outputPcm == NULL) {
        usedGpuMemoryBytes += nFFT * sizeof(hipfftReal);
        goto END;
    }

    NormalizeOutputPcm(pcmSamples);

    // �o��bit depth == 24bit
    meta.bitsPerSample = 24;
    if (!WriteFlacFile(meta, picture, pcmSamples, argv[3])) {
        printf("Error: WriteFlac(%S) failed\n", argv[3]);
        goto END;
    }

    result = 0;

END:
    delete [] picture;
    picture = NULL;

    for (size_t i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].Term();
    }
    pcmSamples.clear();

    crossfeedParam.Term();

    if (result != 0) {
        printf("Failed!\n");
    } else {
        printf("Used GPU memory: %lld Mbytes.\n", usedGpuMemoryBytes/1024/1024);
        printf("Succeeded to write %S.\n", argv[3]);
    }

    return result;
}