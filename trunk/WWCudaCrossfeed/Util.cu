#include "Util.h"
#include <stdio.h>

int64_t gCudaAllocatedBytes = 0;
int64_t gCudaMaxBytes = 0;

size_t
NextPowerOf2(size_t v)
{
    size_t result = 1;
    if (INT_MAX+1U < v) {
        printf("Error: NextPowerOf2(%d) too large!\n", v);
        return 0;
    }
    while (result < v) {
        result *= 2;
    }
    return result;
}

bool
ReadOneLine(FILE *fp, char *line_return, size_t lineBytes)
{
    line_return[0] = 0;
    int c;
    int pos = 0;

    do {
        c = fgetc(fp);
        if (c == EOF || c == '\n') {
            break;
        }

        if (c != '\r') {
            line_return[pos] = (char)c;
            line_return[pos+1] = 0;
            ++pos;
        }
    } while (c != EOF && pos < (int)lineBytes -1);

    return c != EOF;
}

void
GetBestBlockThreadSize(int count, dim3 &threads_return, dim3 &blocks_return)
{
    if ((count / WW_NUM_THREADS_PER_BLOCK) <= 1) {
        threads_return.x = count;
    } else {
        threads_return.x = WW_NUM_THREADS_PER_BLOCK;
        threads_return.y = 1;
        threads_return.z = 1;
        int countRemain = count / WW_NUM_THREADS_PER_BLOCK;
        if ((countRemain / WW_BLOCK_X) <= 1) {
            blocks_return.x = countRemain;
            blocks_return.y = 1;
            blocks_return.z = 1;
        } else {
            blocks_return.x = WW_BLOCK_X;
            countRemain /= WW_BLOCK_X;
            blocks_return.y = countRemain;
            blocks_return.z = 1;
        }
    }
}

void
CrossfeedParam::Term(void) {
    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        delete [] coeffs[i];
        coeffs[i] = NULL;

        CHK_CUDAFREE(spectra[i], fftSize * sizeof(hipfftComplex));
    }
}

void
PcmSamplesPerChannel::Term(void)
{
    delete [] inputPcm;
    inputPcm = NULL;

    delete [] outputPcm;
    outputPcm = NULL;

    CHK_CUDAFREE(spectrum, fftSize * sizeof(hipfftComplex));
}

