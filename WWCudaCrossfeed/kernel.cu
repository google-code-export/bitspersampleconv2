// 日本語

#include "Util.h"
#include "CrossfeedF.h"
#include <assert.h>

int wmain(int argc, wchar_t *argv[])
{
    if (argc != 4) {
        printf("Usage: %S coeffFile inputFile outputFile\n", argv[0]);
        return 1;
    }

    const wchar_t *coeffPath = argv[1];
    const wchar_t *fromPath = argv[2];
    const wchar_t *toPath = argv[3];

    return WWRunCrossfeedF(coeffPath, fromPath, toPath);
}