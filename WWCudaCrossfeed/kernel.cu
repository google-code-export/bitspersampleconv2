#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h> //< memset()
#include <math.h>
#include <assert.h>

#include <hipfft/hipfft.h>

#include "WWFlacRW.h"
#include <vector>
#include <float.h>

#define CROSSFEED_COEF_NUM (4)
#define NUM_THREADS_PER_BLOCK (32)
#define BLOCK_X (32768)

struct CrossfeedParam {
    int numChannels;
    float *coeffs[CROSSFEED_COEF_NUM];
    hipfftComplex *spectra[CROSSFEED_COEF_NUM];

    int sampleRate;
    int coeffSize;

    CrossfeedParam(void) {
        numChannels = 0;
        sampleRate = 0;
        coeffSize = 0;

        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            coeffs[i]  = NULL;
            spectra[i] = NULL;
        }
    }

    void Term(void) {
        for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
            delete [] coeffs[i];
            coeffs[i] = NULL;

            hipFree(spectra[i]);
            spectra[i] = NULL;
        }
    }
};

struct PcmSamplesPerChannel {
    size_t totalSamples;
    float *inputSamples;
    float *outputSamples;
    hipfftComplex *spectrum;

    void Init(void) {
        inputSamples = NULL;
        outputSamples = NULL;
        spectrum = NULL;
    }

    void Term(void) {
        delete [] inputSamples;
        inputSamples = NULL;

        delete [] outputSamples;
        outputSamples = NULL;

        hipFree(spectrum);
        spectrum = NULL;
    }
};

static bool
ReadOneLine(FILE *fp, char *line_return, size_t lineBytes)
{
    line_return[0] = 0;
    int c;
    int pos = 0;

    do {
        c = fgetc(fp);
        if (c == EOF || c == '\n') {
            break;
        }

        if (c != '\r') {
            line_return[pos] = (char)c;
            line_return[pos+1] = 0;
            ++pos;
        }
    } while (c != EOF && pos < (int)lineBytes -1);

    return c != EOF;
}

#define CHECKED(x) if (!(x)) { goto END; }

static bool
ReadCrossfeeedParamsFromFile(const wchar_t *path, CrossfeedParam *param_return)
{
    assert(param_return);

    char buff[256];
    bool result = false;
    FILE *fp;
    errno_t ercd = _wfopen_s(&fp, path, L"rb");
    if (NULL == fp || 0 != ercd) {
        return false;
    }

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    CHECKED(0 == strncmp(buff, "CFD1", 4));

    param_return->numChannels = 2;

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->sampleRate);

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->coeffSize);

    CHECKED(0 < param_return->coeffSize);

    for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
        param_return->coeffs[ch] = new float[param_return->coeffSize];
    }

    for (int i=0; i<param_return->coeffSize; ++i) {
#if CROSSFEED_COEF_NUM != 4
#  error
#endif
        double v[CROSSFEED_COEF_NUM];

        CHECKED(ReadOneLine(fp, buff, sizeof buff));
        sscanf(buff, "%lf, %lf, %lf, %lf", &v[0], &v[1], &v[2], &v[3]);

        for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
            param_return->coeffs[ch][i] = (float)v[ch];
        }
    }

    result = true;

END:
    fclose(fp);
    fp = NULL;
    return result;
}

static void
SetInputPcmSamples(uint8_t *buff, int bitsPerSample, PcmSamplesPerChannel *ppc_return)
{
    assert(ppc_return);

    switch (bitsPerSample) {
    case 16:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            short v = (short)(buff[samplePos*2] + (buff[samplePos*2+1]<<8));
            ppc_return->inputSamples[samplePos] = float(v) * (1.0f / 32768.0f);
        }
        break;
    case 24:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            int v = (int)((buff[samplePos*3]<<8) + (buff[samplePos*3+1]<<16) + (buff[samplePos*3+2]<<24));
            ppc_return->inputSamples[samplePos] = float(v) * (1.0f / 2147483648.0f);
        }
        break;
    default:
        assert(!"not supported");
        break;
    }
}

static size_t
NextPowerOf2(size_t v)
{
    size_t result = 1;
    if (INT_MAX+1U < v) {
        printf("Error: NextPowerOf2(%d) too large!\n", v);
        return 0;
    }
    while (result < v) {
        result *= 2;
    }
    return result;
}
static const char *
CudaFftGetErrorString(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS:       return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:  return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:  return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:  return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:    return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:   return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:   return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";

        case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE:            return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR:               return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE:              return "HIPFFT_NO_WORKSPACE";
        default: return "unknown";
    }
}


#define CHK_CUDAERROR(x)                                                              \
    ercd = x;                                                                         \
    if (hipSuccess != ercd) {                                                        \
        printf("%s failed. errorcode=%d (%s)\n", #x, ercd, hipGetErrorString(ercd)); \
        return NULL;                                                                  \
    }

#define CHK_CUFFT(x)                                                                               \
    fftResult = x;                                                                                 \
    if (hipSuccess != fftResult) {                                                                \
        printf("%s failed. errorcode=%d (%s)\n", #x, fftResult, CudaFftGetErrorString(fftResult)); \
        return NULL;                                                                               \
    }

static hipfftComplex *
CreateSpectrum(float *timeDomainData, int numSamples, int fftSize)
{
    hipfftReal *cuFromT = NULL;
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftComplex *spectrum;
    hipfftHandle plan = 0;

    CHK_CUDAERROR(hipMalloc((void**)&cuFromT, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemset((void*)cuFromT, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(cuFromT, timeDomainData, numSamples * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAERROR(hipMalloc((void**)&spectrum, sizeof(hipfftComplex)*fftSize));

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, cuFromT, spectrum));

    hipDeviceSynchronize();

    if (plan != 0) {
        hipfftDestroy(plan);
        plan = 0;
    }

    hipFree(cuFromT);
    cuFromT = NULL;

    return spectrum;
}

__global__ void
ElementWiseMulCuda(hipfftComplex *C, hipfftComplex *A, hipfftComplex *B)
{
    int offs = threadIdx.x + (blockDim.x * blockDim.y) * (blockIdx.x + gridDim.x * blockIdx.y);
    C[offs].x = A[offs].x * B[offs].x - A[offs].y * B[offs].y;
    C[offs].y = A[offs].x * B[offs].y + A[offs].y * B[offs].x;
}

__global__ void
ElementWiseAddCuda(hipfftReal *C, hipfftReal *A, hipfftReal *B)
{
    int offs = threadIdx.x + NUM_THREADS_PER_BLOCK * (blockIdx.x + BLOCK_X * blockIdx.y);
    C[offs] = A[offs] + B[offs];
}

static float *
CrossfeedMix(hipfftComplex *inPcm[2], hipfftComplex *coeff[2], int nFFT, int pcmSamples)
{
    dim3 threads(1);
    dim3 blocks(1);
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftHandle plan = 0;
    hipfftComplex *cuFreq = NULL;
    hipfftReal *cuTime[2] = {NULL, NULL};
    hipfftReal *cuTimeMixed = NULL;

    if ((nFFT / NUM_THREADS_PER_BLOCK) <= 1) {
        threads.x = nFFT;
    } else {
        threads.x = NUM_THREADS_PER_BLOCK;
        threads.y = 1;
        threads.z = 1;
        int countRemain = nFFT / NUM_THREADS_PER_BLOCK;
        if ((countRemain / BLOCK_X) <= 1) {
            blocks.x = countRemain;
            blocks.y = 1;
            blocks.z = 1;
        } else {
            blocks.x = BLOCK_X;
            countRemain /= BLOCK_X;
            blocks.y = countRemain;
            blocks.z = 1;
        }
    }

    CHK_CUDAERROR(hipMalloc((void**)&cuFreq,      sizeof(hipfftComplex)*nFFT));
    CHK_CUDAERROR(hipMalloc((void**)&cuTime[0],   sizeof(hipfftReal)*nFFT));
    CHK_CUDAERROR(hipMalloc((void**)&cuTime[1],   sizeof(hipfftReal)*nFFT));

    hipDeviceSynchronize();

    for (int ch=0; ch<2; ++ch) {
        ElementWiseMulCuda<<<blocks, threads>>>(cuFreq, inPcm[ch], coeff[ch]);
    
        hipDeviceSynchronize();

        CHK_CUFFT(hipfftPlan1d(&plan, nFFT, HIPFFT_C2R, 1));
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch]));

        hipDeviceSynchronize();

        hipfftDestroy(plan);
        plan = 0;
    }

    hipFree(cuFreq);
    cuFreq = NULL;

    CHK_CUDAERROR(hipMalloc((void**)&cuTimeMixed, sizeof(hipfftReal)*nFFT));

    hipDeviceSynchronize();

    ElementWiseAddCuda<<<blocks, threads>>>(cuTimeMixed, cuTime[0], cuTime[1]);

    hipDeviceSynchronize();

    for (int ch=0; ch<2; ++ch) {
        hipFree(cuTime[ch]);
        cuTime[ch] = NULL;
    }

    hipDeviceSynchronize();

    float *result = new float[pcmSamples];
    CHK_CUDAERROR(hipMemcpy(result, cuTimeMixed, pcmSamples * sizeof(float), hipMemcpyDeviceToHost));

    hipDeviceSynchronize();

    hipFree(cuTimeMixed);
    cuTimeMixed = NULL;

    hipDeviceSynchronize();

    return result;
}

static void
NormalizeOutputPcm(std::vector<PcmSamplesPerChannel> & pcmSamples)
{
    float minV = FLT_MAX;
    float maxV = FLT_MIN;

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            if (maxV < pcmSamples[ch].outputSamples[i]) {
                maxV = pcmSamples[ch].outputSamples[i];
            }
            if (pcmSamples[ch].outputSamples[i] < minV) {
                minV = pcmSamples[ch].outputSamples[i];
            }
        }
    }

    float absMax = (fabsf(minV) < fabsf(maxV)) ? fabsf(maxV) : fabsf(minV);
    float scale = 1.0f;
    if ((8388607.0f / 8388608.0f) < absMax) {
        scale = (8388607.0f / 8388608.0f) / absMax;
    }

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            pcmSamples[ch].outputSamples[i] *= scale;
        }
    }
}

static bool
WriteFlacFile(const WWFlacMetadata &meta, const uint8_t *picture, std::vector<PcmSamplesPerChannel> &pcmSamples, const wchar_t *path)
{
    bool result = false;
    int rv;
    int pictureBytes = meta.pictureBytes;

    int id = WWFlacRW_EncodeInit(meta);
    if (id < 0) {
        return false;
    }

    if (0 < pictureBytes) {
        rv = WWFlacRW_EncodeSetPicture(id, picture, pictureBytes);
        if (rv < 0) {
            goto END;
        }
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        uint8_t *pcmDataUint8 = new uint8_t[(size_t)(meta.totalSamples * 3)];
        for (int i=0; i<meta.totalSamples; ++i) {
            int v = (int)(8388608.0f * pcmSamples[ch].outputSamples[i]);
            pcmDataUint8[i*3+0] = v&0xff;
            pcmDataUint8[i*3+1] = (v>>8)&0xff;
            pcmDataUint8[i*3+2] = (v>>16)&0xff;
        }

        rv = WWFlacRW_EncodeAddPcm(id, ch, pcmDataUint8, meta.totalSamples*3);
        if (rv < 0) {
            goto END;
        }
        delete [] pcmDataUint8;
        pcmDataUint8 = NULL;
    }

    rv = WWFlacRW_EncodeRun(id, path);
    if (rv < 0) {
        goto END;
    }

    result = true;
END:

    WWFlacRW_EncodeEnd(id);

    return result;
}

int wmain(int argc, wchar_t *argv[])
{
    int result = 1;
    int ercd;
    int id = -1;
    size_t nFFT;
    CrossfeedParam crossfeedParam;
    WWFlacMetadata meta;
    uint8_t * picture = NULL;
    hipfftComplex * inPcmSpectra[2];
    int64_t usedGpuMemoryBytes = 0;

    std::vector<PcmSamplesPerChannel> pcmSamples;

    if (argc != 4) {
        printf("Usage: %S coeffFile inputFile outputFile\n", argv[0]);
        goto END;
    }

    if (!ReadCrossfeeedParamsFromFile(argv[1], &crossfeedParam)) {
        printf("Error: could not read crossfeed param file %S\n", argv[1]);
        goto END;
    }

    id = WWFlacRW_DecodeAll(argv[2]);
    if (id < 0) {
        printf("Error: Read failed %S\n", argv[2]);
        goto END;
    }

    ercd = WWFlacRW_GetDecodedMetadata(id, meta);
    if (ercd < 0) {
        printf("Error: Read meta failed %S\n", argv[2]);
        goto END;
    }

    if (0 < meta.pictureBytes) {
        picture = new uint8_t[meta.pictureBytes];
        ercd = WWFlacRW_GetDecodedPicture(id, picture, meta.pictureBytes);
        if (ercd < 0) {
            printf("Error: Read meta failed %S\n", argv[2]);
            goto END;
        }
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: channel count mismatch. FLAC ch=%d, crossfeed ch=%d\n", meta.channels, crossfeedParam.numChannels);
        goto END;
    }

    if (meta.channels != crossfeedParam.numChannels) {
        printf("Error: samplerate mismatch. FLAC=%d, crossfeed=%d\n", meta.sampleRate, crossfeedParam.sampleRate);
        goto END;
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        size_t bytes = (size_t)(meta.totalSamples * (meta.bitsPerSample/8));
        uint8_t *buff = new uint8_t[bytes];
        WWFlacRW_GetDecodedPcmBytes(id, ch, 0, buff, bytes);

        PcmSamplesPerChannel ppc;
        ppc.Init();
        ppc.totalSamples = (size_t)meta.totalSamples;
        ppc.inputSamples = new float[(size_t)(meta.totalSamples * sizeof(float))];
        ppc.outputSamples = NULL;
        ppc.spectrum = NULL;

        SetInputPcmSamples(buff, meta.bitsPerSample, &ppc);

        delete [] buff;
        buff = NULL;

        pcmSamples.push_back(ppc);
    }

    WWFlacRW_DecodeEnd(id);
    id = -1;

    nFFT = (size_t)((crossfeedParam.coeffSize < meta.totalSamples) ? meta.totalSamples : crossfeedParam.coeffSize);
    nFFT = NextPowerOf2(nFFT);

    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        crossfeedParam.spectra[i] = CreateSpectrum(crossfeedParam.coeffs[i], crossfeedParam.coeffSize, nFFT);
        if (crossfeedParam.spectra[i] == NULL) {
            goto END;
        }
        usedGpuMemoryBytes += nFFT * sizeof(hipfftComplex);
    }
    for (int ch=0; ch<meta.channels; ++ch) {
        pcmSamples[ch].spectrum = CreateSpectrum(pcmSamples[ch].inputSamples, pcmSamples[ch].totalSamples, nFFT);
        if (pcmSamples[ch].spectrum == NULL) {
            goto END;
        }
        usedGpuMemoryBytes += nFFT * sizeof(hipfftComplex);
    }

    inPcmSpectra[0] = pcmSamples[0].spectrum;
    inPcmSpectra[1] = pcmSamples[1].spectrum;
    pcmSamples[0].outputSamples = CrossfeedMix(inPcmSpectra, &crossfeedParam.spectra[0], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[0].outputSamples == NULL) {
        usedGpuMemoryBytes += nFFT * sizeof(hipfftReal);
        goto END;
    }
    pcmSamples[1].outputSamples = CrossfeedMix(inPcmSpectra, &crossfeedParam.spectra[2], nFFT, pcmSamples[0].totalSamples);
    if (pcmSamples[1].outputSamples == NULL) {
        usedGpuMemoryBytes += nFFT * sizeof(hipfftReal);
        goto END;
    }

    NormalizeOutputPcm(pcmSamples);

    // �o��bit depth == 24bit
    meta.bitsPerSample = 24;
    if (!WriteFlacFile(meta, picture, pcmSamples, argv[3])) {
        printf("Error: WriteFlac(%S) failed\n", argv[3]);
        goto END;
    }

    result = 0;

END:
    delete [] picture;
    picture = NULL;

    for (size_t i=0; i<pcmSamples.size(); ++i) {
        pcmSamples[i].Term();
    }
    pcmSamples.clear();

    crossfeedParam.Term();

    if (result != 0) {
        printf("Failed!\n");
    } else {
        printf("Used GPU memory: %lld Mbytes.\n", usedGpuMemoryBytes/1024/1024);
        printf("Succeeded to write %S.\n", argv[3]);
    }

    return result;
}