#include "Util.h"
#include <stdio.h>

int64_t gCudaAllocatedBytes = 0;
int64_t gCudaMaxBytes = 0;



// 44.1kHz�p 1kHz�ȉ������o��LPF�B
float gLpf[WW_CROSSOVER_COEFF_LENGTH] = {
        0.005228327, 0.003249754, 0.004192373, 0.005265026,
        0.006468574, 0.007797099, 0.009237486, 0.010779043,
        0.012417001, 0.014132141, 0.01589555, 0.017701121,
        0.019508703, 0.021304869, 0.023059883,0.024747905,
        0.02634363, 0.027823228, 0.029158971, 0.030331066,
        0.031319484, 0.032104039, 0.032676435, 0.033022636,
        0.033138738, 0.033022636, 0.032676435, 0.032104039,
        0.031319484, 0.030331066, 0.029158971, 0.027823228,
        0.02634363, 0.024747905, 0.023059883, 0.021304869,
        0.019508703, 0.017701121, 0.01589555, 0.014132141,
        0.012417001, 0.010779043, 0.009237486, 0.007797099,
        0.006468574, 0.005265026, 0.004192373, 0.003249754,
        0.005228327 };

// 44.1kHz�p 1kHz�ȏ�����o��HPF�BLPF�ƃR���v�������^���[�ɂȂ��Ă���B
float gHpf[WW_CROSSOVER_COEFF_LENGTH] = {
        -0.005228327,-0.003249754,-0.004192373,-0.005265026,
        -0.006468574,-0.007797099,-0.009237486,-0.010779043,
        -0.012417001,-0.014132141,-0.01589555,-0.017701121,
        -0.019508703,-0.021304869,-0.023059883,-0.024747905,
        -0.02634363,-0.027823228,-0.029158971,-0.030331066,
        -0.031319484,-0.032104039,-0.032676435,-0.033022636,
        0.966861262,-0.033022636,-0.032676435,-0.032104039,
        -0.031319484,-0.030331066,-0.029158971,-0.027823228,
        -0.02634363,-0.024747905,-0.023059883,-0.021304869,
        -0.019508703,-0.017701121,-0.01589555,-0.014132141,
        -0.012417001,-0.010779043,-0.009237486,-0.007797099,
        -0.006468574,-0.005265026,-0.004192373,-0.003249754,
        -0.005228327};

size_t
NextPowerOf2(size_t v)
{
    size_t result = 1;
    if (INT_MAX+1U < v) {
        printf("Error: NextPowerOf2(%d) too large!\n", v);
        return 0;
    }
    while (result < v) {
        result *= 2;
    }
    return result;
}

bool
ReadOneLine(FILE *fp, char *line_return, size_t lineBytes)
{
    line_return[0] = 0;
    int c;
    int pos = 0;

    do {
        c = fgetc(fp);
        if (c == EOF || c == '\n') {
            break;
        }

        if (c != '\r') {
            line_return[pos] = (char)c;
            line_return[pos+1] = 0;
            ++pos;
        }
    } while (c != EOF && pos < (int)lineBytes -1);

    return c != EOF;
}

void
GetBestBlockThreadSize(int count, dim3 &threads_return, dim3 &blocks_return)
{
    if ((count / WW_NUM_THREADS_PER_BLOCK) <= 1) {
        threads_return.x = count;
    } else {
        threads_return.x = WW_NUM_THREADS_PER_BLOCK;
        threads_return.y = 1;
        threads_return.z = 1;
        int countRemain = count / WW_NUM_THREADS_PER_BLOCK;
        if ((countRemain / WW_BLOCK_X) <= 1) {
            blocks_return.x = countRemain;
            blocks_return.y = 1;
            blocks_return.z = 1;
        } else {
            blocks_return.x = WW_BLOCK_X;
            countRemain /= WW_BLOCK_X;
            blocks_return.y = countRemain;
            blocks_return.z = 1;
        }
    }
}

void
CrossfeedParam::Term(void) {
    for (int i=0; i<CROSSFEED_COEF_NUM; ++i) {
        delete [] coeffs[i];
        coeffs[i] = NULL;

        CHK_CUDAFREE(spectra[i], fftSize * sizeof(hipfftComplex));
    }
}

void
PcmSamplesPerChannel::Term(void)
{
    delete [] inputPcm;
    inputPcm = NULL;

    delete [] outputPcm;
    outputPcm = NULL;

    CHK_CUDAFREE(spectrum, fftSize * sizeof(hipfftComplex));
}

const char *
CudaFftGetErrorString(hipfftResult error)
{
    switch (error) {
        case HIPFFT_SUCCESS:       return "HIPFFT_SUCCESS";
        case HIPFFT_INVALID_PLAN:  return "HIPFFT_INVALID_PLAN";
        case HIPFFT_ALLOC_FAILED:  return "HIPFFT_ALLOC_FAILED";
        case HIPFFT_INVALID_TYPE:  return "HIPFFT_INVALID_TYPE";
        case HIPFFT_INVALID_VALUE: return "HIPFFT_INVALID_VALUE";

        case HIPFFT_INTERNAL_ERROR: return "HIPFFT_INTERNAL_ERROR";
        case HIPFFT_EXEC_FAILED:    return "HIPFFT_EXEC_FAILED";
        case HIPFFT_SETUP_FAILED:   return "HIPFFT_SETUP_FAILED";
        case HIPFFT_INVALID_SIZE:   return "HIPFFT_INVALID_SIZE";
        case HIPFFT_UNALIGNED_DATA: return "HIPFFT_UNALIGNED_DATA";

        case HIPFFT_INCOMPLETE_PARAMETER_LIST: return "HIPFFT_INCOMPLETE_PARAMETER_LIST";
        case HIPFFT_INVALID_DEVICE:            return "HIPFFT_INVALID_DEVICE";
        case HIPFFT_PARSE_ERROR:               return "HIPFFT_PARSE_ERROR";
        case HIPFFT_NO_WORKSPACE:              return "HIPFFT_NO_WORKSPACE";
        default: return "unknown";
    }
}