#include "hip/hip_runtime.h"
#include "CrossfeedF.h"

#include <assert.h>
#include "WWFlacRW.h"

bool
ReadCrossfeeedParamsFromFileF(const wchar_t *path, CrossfeedParam *param_return)
{
    assert(param_return);

    char buff[256];
    bool result = false;
    FILE *fp;
    errno_t ercd = _wfopen_s(&fp, path, L"rb");
    if (NULL == fp || 0 != ercd) {
        return false;
    }

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    CHECKED(0 == strncmp(buff, "CFD2", 4));

    param_return->numChannels = 2;

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->sampleRate);

    CHECKED(ReadOneLine(fp, buff, sizeof buff));
    sscanf(buff, "%d", &param_return->coeffSize);

    CHECKED(0 < param_return->coeffSize);

    // �R�����g�s�B�X�L�b�v����B
    CHECKED(ReadOneLine(fp, buff, sizeof buff));

    for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
        param_return->coeffs[ch] = new float[param_return->coeffSize];
    }

    for (int i=0; i<param_return->coeffSize; ++i) {
#if CROSSFEED_COEF_NUM != 8
#  error
#endif
        double v[CROSSFEED_COEF_NUM];

        CHECKED(ReadOneLine(fp, buff, sizeof buff));
        CHECKED(8 == sscanf(buff, "%lf, %lf, %lf, %lf, %lf, %lf, %lf, %lf",
                &v[0], &v[1], &v[2], &v[3], &v[4], &v[5], &v[6], &v[7]));

        for (int ch=0; ch<CROSSFEED_COEF_NUM; ++ch) {
            param_return->coeffs[ch][i] = (float)v[ch];
        }
    }

    result = true;

END:
    fclose(fp);
    fp = NULL;
    return result;
}

void
SetInputPcmSamplesF(uint8_t *buff, int bitsPerSample, PcmSamplesPerChannel *ppc_return)
{
    assert(ppc_return);

    switch (bitsPerSample) {
    case 16:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            short v = (short)(buff[samplePos*2] + (buff[samplePos*2+1]<<8));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 32768.0f);
        }
        break;
    case 24:
        for (size_t samplePos=0; samplePos<ppc_return->totalSamples; ++samplePos) {
            int v = (int)((buff[samplePos*3]<<8) + (buff[samplePos*3+1]<<16) + (buff[samplePos*3+2]<<24));
            ppc_return->inputPcm[samplePos] = float(v) * (1.0f / 2147483648.0f);
        }
        break;
    default:
        assert(!"not supported");
        break;
    }
}

__global__ void
ElementWiseMulCudaF(hipfftComplex *C, hipfftComplex *A, hipfftComplex *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs].x = A[offs].x * B[offs].x - A[offs].y * B[offs].y;
    C[offs].y = A[offs].x * B[offs].y + A[offs].y * B[offs].x;
}

__global__ void
ElementWiseAddCudaF(hipfftReal *C, hipfftReal *A, hipfftReal *B)
{
    int offs = threadIdx.x + WW_NUM_THREADS_PER_BLOCK * (blockIdx.x + WW_BLOCK_X * blockIdx.y);
    C[offs] = A[offs] + B[offs];
}

static void
CudaElementWiseMulF(int count, hipfftComplex *dest, hipfftComplex *from0, hipfftComplex *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseMulCudaF<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

static void
CudaElementWiseAddF(int count, hipfftReal *dest, hipfftReal *from0, hipfftReal *from1)
{
    dim3 threads(1);
    dim3 blocks(1);

    GetBestBlockThreadSize(count, threads, blocks);
    hipDeviceSynchronize();
    ElementWiseAddCudaF<<<blocks, threads>>>(dest, from0, from1);
    hipDeviceSynchronize();
}

hipfftComplex *
CreateSpectrumF(float *timeDomainData, int numSamples, int fftSize)
{
    hipfftReal *cuFromT = NULL;
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftComplex *spectrum;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&cuFromT, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)cuFromT, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(cuFromT, timeDomainData, numSamples * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&spectrum, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, cuFromT, spectrum));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFromT, sizeof(hipfftReal)*fftSize);
    return spectrum;
}

float *
FirFilterF(float *firCoeff, size_t firCoeffNum, PcmSamplesPerChannel &input, PcmSamplesPerChannel *pOutput)
{
    size_t fftSize = (firCoeffNum < input.totalSamples) ? input.totalSamples: firCoeffNum;
    fftSize = NextPowerOf2(fftSize);
    if (fftSize == 0) {
        return NULL;
    }

    hipError_t ercd;
    hipfftResult fftResult;
    hipfftReal *coefTime = NULL;
    hipfftReal *pcmTime = NULL;
    hipfftReal *resultTime = NULL;
    hipfftComplex *coefFreq = NULL;
    hipfftComplex *pcmFreq = NULL;
    hipfftComplex *resultFreq = NULL;
    hipfftHandle plan = 0;

    CHK_CUDAMALLOC((void**)&coefTime, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)coefTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(coefTime, firCoeff, firCoeffNum * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&coefFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_R2C, 1));
    CHK_CUFFT(hipfftExecR2C(plan, coefTime, coefFreq));

    CHK_CUDAFREE(coefTime, sizeof(hipfftReal)*fftSize);

    CHK_CUDAMALLOC((void**)&pcmTime, sizeof(hipfftReal)*fftSize);
    CHK_CUDAERROR(hipMemset((void*)pcmTime, 0, sizeof(hipfftReal)*fftSize));
    CHK_CUDAERROR(hipMemcpy(pcmTime, input.inputPcm, input.totalSamples * sizeof(float), hipMemcpyHostToDevice));
    CHK_CUDAMALLOC((void**)&pcmFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUFFT(hipfftExecR2C(plan, pcmTime, pcmFreq));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(pcmTime, sizeof(hipfftReal)*fftSize);

    CHK_CUDAMALLOC((void**)&resultFreq, sizeof(hipfftComplex)*fftSize);
    CudaElementWiseMulF(fftSize, resultFreq, coefFreq, pcmFreq);

    CHK_CUDAFREE(coefFreq, sizeof(hipfftComplex)*fftSize);
    CHK_CUDAFREE(pcmFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUDAMALLOC((void**)&resultTime, sizeof(hipfftReal)*fftSize);

    CHK_CUFFT(hipfftPlan1d(&plan, fftSize, HIPFFT_C2R, 1));
    CHK_CUFFT(hipfftExecC2R(plan, resultFreq, resultTime));

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(resultFreq, sizeof(hipfftComplex)*fftSize);

    CHK_CUDAERROR(hipMemcpy(pOutput->inputPcm, resultTime, input.totalSamples * sizeof(float), hipMemcpyDeviceToHost));
    CHK_CUDAFREE(resultTime, sizeof(hipfftReal)*fftSize);

    return pOutput->inputPcm;
}

float *
CrossfeedMixF(hipfftComplex *inPcmSpectra[PCT_NUM], hipfftComplex *coeffLo[2],
        hipfftComplex *coeffHi[2], int nFFT, int pcmSamples)
{
    hipError_t ercd;
    hipfftResult fftResult;
    hipfftHandle plan = 0;
    hipfftComplex *cuFreq = NULL;
    hipfftReal *cuTime[PCT_NUM] = {NULL, NULL, NULL, NULL};
    hipfftReal *cuTimeMixedLo = NULL;
    hipfftReal *cuTimeMixedHi = NULL;
    hipfftReal *cuTimeMixed = NULL;

    CHK_CUDAMALLOC((void**)&cuFreq, sizeof(hipfftComplex)*nFFT);
    CHK_CUFFT(hipfftPlan1d(&plan, nFFT, HIPFFT_C2R, 1));

    for (int ch=0; ch<2; ++ch) {
        CudaElementWiseMulF(nFFT, cuFreq, inPcmSpectra[ch*2], coeffLo[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2], sizeof(hipfftReal)*nFFT);
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2]));

        CudaElementWiseMulF(nFFT, cuFreq, inPcmSpectra[ch*2+1], coeffHi[ch]);

        CHK_CUDAMALLOC((void**)&cuTime[ch*2+1], sizeof(hipfftReal)*nFFT);
        CHK_CUFFT(hipfftExecC2R(plan, cuFreq, cuTime[ch*2+1]));
    }

    hipfftDestroy(plan);
    plan = 0;

    CHK_CUDAFREE(cuFreq, sizeof(hipfftComplex)*nFFT);

    CHK_CUDAMALLOC((void**)&cuTimeMixedLo, sizeof(hipfftReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixedHi, sizeof(hipfftReal)*nFFT);
    CHK_CUDAMALLOC((void**)&cuTimeMixed, sizeof(hipfftReal)*nFFT);

    CudaElementWiseAddF(nFFT, cuTimeMixedLo, cuTime[0], cuTime[2]);
    CudaElementWiseAddF(nFFT, cuTimeMixedHi, cuTime[1], cuTime[3]);
    CudaElementWiseAddF(nFFT, cuTimeMixed, cuTimeMixedLo, cuTimeMixedHi);

    for (int i=0; i<PCT_NUM; ++i) {
        CHK_CUDAFREE(cuTime[i], sizeof(hipfftReal)*nFFT);
    }
    CHK_CUDAFREE(cuTimeMixedLo, sizeof(hipfftReal)*nFFT);
    CHK_CUDAFREE(cuTimeMixedHi, sizeof(hipfftReal)*nFFT);

    float *result = new float[pcmSamples];
    CHK_CUDAERROR(hipMemcpy(result, cuTimeMixed, pcmSamples * sizeof(float), hipMemcpyDeviceToHost));

    CHK_CUDAFREE(cuTimeMixed, sizeof(hipfftReal)*nFFT);

    return result;
}

void
NormalizeOutputPcmF(std::vector<PcmSamplesPerChannel> & pcmSamples)
{
    float minV = FLT_MAX;
    float maxV = FLT_MIN;

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }

        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            if (maxV < pcmSamples[ch].outputPcm[i]) {
                maxV = pcmSamples[ch].outputPcm[i];
            }
            if (pcmSamples[ch].outputPcm[i] < minV) {
                minV = pcmSamples[ch].outputPcm[i];
            }
        }
    }

    float absMax = (fabsf(minV) < fabsf(maxV)) ? fabsf(maxV) : fabsf(minV);
    float scale = 1.0f;
    if ((8388607.0f / 8388608.0f) < absMax) {
        scale = (8388607.0f / 8388608.0f) / absMax;
    }

    for (size_t ch=0; ch<pcmSamples.size(); ++ch) {
        if (pcmSamples[ch].outputPcm == NULL) {
            continue;
        }
        for (size_t i=0; i<pcmSamples[ch].totalSamples; ++i) {
            pcmSamples[ch].outputPcm[i] *= scale;
        }
    }
}

bool
WriteFlacFileF(const WWFlacMetadata &meta, const uint8_t *picture,
        std::vector<PcmSamplesPerChannel> &pcmSamples, const wchar_t *path)
{
    bool result = false;
    int rv;
    int pictureBytes = meta.pictureBytes;

    int id = WWFlacRW_EncodeInit(meta);
    if (id < 0) {
        return false;
    }

    if (0 < pictureBytes) {
        rv = WWFlacRW_EncodeSetPicture(id, picture, pictureBytes);
        if (rv < 0) {
            goto END;
        }
    }

    for (int ch=0; ch<meta.channels; ++ch) {
        uint8_t *pcmDataUint8 = new uint8_t[(size_t)(meta.totalSamples * 3)];
        for (int i=0; i<meta.totalSamples; ++i) {
            int v = (int)(8388608.0f * pcmSamples[ch].outputPcm[i]);
            pcmDataUint8[i*3+0] = v&0xff;
            pcmDataUint8[i*3+1] = (v>>8)&0xff;
            pcmDataUint8[i*3+2] = (v>>16)&0xff;
        }

        rv = WWFlacRW_EncodeAddPcm(id, ch, pcmDataUint8, meta.totalSamples*3);
        if (rv < 0) {
            goto END;
        }
        delete [] pcmDataUint8;
        pcmDataUint8 = NULL;
    }

    rv = WWFlacRW_EncodeRun(id, path);
    if (rv < 0) {
        goto END;
    }

    result = true;
END:

    WWFlacRW_EncodeEnd(id);
    return result;
}